#include "hip/hip_runtime.h"
#include "Matrix.h"
#include <iostream>
#include <complex>
#include <chrono>
#include <random>
#include <fstream>
#include <omp.h>

using Time = std::chrono::high_resolution_clock;
using fsec = std::chrono::duration<float>;

void gen_random(Matrix<int> & m,    const int min, const int max);
void gen_random(Matrix<double> & m, const int min, const int max);


//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

int main(){

    omp_set_num_threads(omp_get_thread_num());
  
    // rozmiar macierzy
    const int size_M = 3;

    Matrix<double> A (size_M, size_M, 0.0);   gen_random(A, -5, 5);
    Matrix<double> B (size_M, size_M, 0.0);   gen_random(B, -5, 5);

    // Wynik macierzy na CPU i zmierzenie czasu
    auto start_CPU = Time::now();
    Matrix<double> C = A*B + 6*A.transpose() + A - 3*B;
    auto end_CPU = Time::now();

    A.set_gpu(true);
    B.set_gpu(true);

    // Wynik macierzy na GPU i zmierzenie czasu
    auto start_GPU = Time::now();
    Matrix<double> D = A*B + 6*A.transpose() + A - 3*B;
    auto end_GPU = Time::now();

    fsec fs1 = end_CPU - start_CPU;
    fsec fs2 = end_GPU - start_GPU;

    std::cout << " * MACIERZ A *" << std::endl;
    std::cout << A ;
    std::cout << " * MACIERZ B *" << std::endl;
    std::cout << B ;
    std::cout << " * MACIERZ C *" << std::endl;
    std::cout << C ;
    std::cout << " * MACIERZ D *" << std::endl;
    std::cout << D ;
    std::cout << "[CPU] Czas macierzy C: " << std::setprecision(16) << fs1.count() << "s" << std::endl;
    std::cout << "[GPU] Czas macierzy D: " << std::setprecision(16) << fs2.count() << "s" << std::endl;

    return 0;
}

//////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////

void gen_random(Matrix<int> & m, const int min, const int max){
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<int> distrib(min, max);

    int r = m.get_row();
    int c = m.get_col();

    for(int i=0; i<r; i++){
        for(int j=0; j<c; j++){
            m.set_element_at(i,j, distrib(gen));
        }
    }
}

void gen_random(Matrix<double> & m, const int min, const int max){
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> distrib(min, max);

    int r = m.get_row();
    int c = m.get_col();

    for(int i=0; i<r; i++){
        for(int j=0; j<c; j++){
            m.set_element_at(i,j, distrib(gen));
        }
    }
}
