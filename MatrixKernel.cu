#include "hip/hip_runtime.h"
#include "MatrixKernel.cuh"


/////////////////////////////////////////////////////////////////
// Konstruktory
/////////////////////////////////////////////////////////////////
template<typename T>
MatrixKernel<T>::MatrixKernel(const T * h_v1, const T * h_v2, T * h_v3, const int r, const int c){
    rows = r;
    cols = c;
    size = r*c;

    hipMalloc((void **)&d_v1, size * sizeof(T));
    hipMalloc((void **)&d_v2, size * sizeof(T));
    hipMalloc((void **)&d_v3, size * sizeof(T));

    hipMemcpy(d_v1, h_v1, size * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_v2, h_v2, size * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_v3, h_v3, size * sizeof(T), hipMemcpyHostToDevice);
}


template<typename T>
MatrixKernel<T>::MatrixKernel(const T* h_v1, const T h_s, T* h_v3, const int r, const int c){
    rows = r;
    cols = c;
    size = r*c;

    hipMalloc((void **)&d_v1, size * sizeof(T));
    hipMalloc((void **)&d_s,         sizeof(T));
    hipMalloc((void **)&d_v3, size * sizeof(T));

    hipMemcpy(d_v1, h_v1, size * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_s,  &h_s,        sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_v3, h_v3, size * sizeof(T), hipMemcpyHostToDevice);
}


template<typename T>
MatrixKernel<T>::MatrixKernel(const T * h_v1, T * h_v3, const int r, const int c){
    rows = r;
    cols = c;
    size = r*c;

    hipMalloc((void **)&d_v1, size * sizeof(T));
    hipMalloc((void **)&d_v3, size * sizeof(T));

    hipMemcpy(d_v1, h_v1, size * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_v3, h_v3, size * sizeof(T), hipMemcpyHostToDevice);
}


/////////////////////////////////////////////////////////////////
// Destruktory
/////////////////////////////////////////////////////////////////
template<typename T>
MatrixKernel<T>::~MatrixKernel(){
    hipFree(d_v1);
    hipFree(d_v2);
    hipFree(d_v3);
    hipFree(d_s);
}


/////////////////////////////////////////////////////////////////
// Kernel - dodawania macierzy
/////////////////////////////////////////////////////////////////
template<typename T>
__global__ void addmKernel(const T* d_v1, const T* d_v2, T* d_v3, size_t size){
    int index = blockIdx.x * 1024 + threadIdx.y * 32 + threadIdx.x;
    if(index < size){
        d_v3[index] = d_v1[index] + d_v2[index];
    }
}

template<typename T>
__host__ void MatrixKernel<T>::addmMatrixKernel(){
    int dbx = 32;
    int dby = 32;
    int dgx = static_cast<int>(this->cols/dbx)+1;
    int dgy = static_cast<int>(this->rows/dby)+1;

    dim3 dimBlock(dbx, dby);
    dim3 dimGrid (dgx, dgy);
    addmKernel<<<dimGrid, dimBlock>>>(this->d_v1, this->d_v2, this->d_v3, this->size);
}


/////////////////////////////////////////////////////////////////
// Kernel - odejmowania macierzy
/////////////////////////////////////////////////////////////////
template<typename T>
__global__ void submKernel(const T* d_v1, const T* d_v2, T* d_v3, size_t size){
    int index = blockIdx.x * 1024 + threadIdx.y * 32 + threadIdx.x;
    if(index < size){
        d_v3[index] = d_v1[index] - d_v2[index];
    }
}


template<typename T>
__host__ void MatrixKernel<T>::submMatrixKernel(){
    int dbx = 32;
    int dby = 32;
    int dgx = static_cast<int>(this->cols/dbx)+1;
    int dgy = static_cast<int>(this->rows/dby)+1;

    dim3 dimBlock(dbx, dby);
    dim3 dimGrid (dgx, dgy);
    submKernel<<<dimGrid, dimBlock>>>(this->d_v1, this->d_v2, this->d_v3, this->size);
}

/////////////////////////////////////////////////////////////////
// funkcja mnozenia macierzy z pamiecia shared
/////////////////////////////////////////////////////////////////

// template<typename T>
// __global__ void mulmKernelSHARED(const T* d_v1, const T* d_v2, T* d_v3, const int r, const int c){
    
//     const int BLOCK_SIZE = 32;

//     int blockRow = blockIdx.y;
//     int blockCol = blockIdx.x;

//     T * Csub = &d_v3[r * BLOCK_SIZE * blockRow + BLOCK_SIZE * blockCol];
    
//     double Cvalue = 0;

//     int row = threadIdx.y;
//     int col = threadIdx.x;

//     int n = static_cast<int>(r/BLOCK_SIZE)+1;
//     for(int i=0; i<n; i++){
        
//         T * Asub = &d_v1[r * BLOCK_SIZE * blockRow + BLOCK_SIZE * i];
//         T * Bsub = &d_v2[r * BLOCK_SIZE * i + BLOCK_SIZE * blockCol];

//         __shared__ T As[BLOCK_SIZE][BLOCK_SIZE];
//         As[row][col] = Asub[row*r + col];

//         __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];
//         Bs[row][col] = Bsub[row*r + col];

//         __syncthreads();

//         for(int j=0; j < BLOCK_SIZE; j++){
//             Cvalue += As[row][j] * Bs[j][col];
//         }

//         __syncthreads();

//         Csub[row*r + col] = Cvalue;
//     }
// }

/////////////////////////////////////////////////////////////////
// Kernel - mnozenia macierzy
/////////////////////////////////////////////////////////////////
template<typename T>
__global__ void mulmKernel(const T* d_v1, const T* d_v2, T* d_v3, const int r, const int c){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < r && col < c){
        for(int i=0; i<r; ++i){
            d_v3[row * r + col] += d_v1[row * r + i] * d_v2[i * r + col];
        }
    }
}


template<typename T>
__host__ void MatrixKernel<T>::mulmMatrixKernel(){
    int dbx = 32;
    int dby = 32;
    int dgx = static_cast<int>(this->cols/dbx)+1;
    int dgy = static_cast<int>(this->rows/dby)+1;    

    dim3 dimBlock(dbx, dby);
    dim3 dimGrid (dgx, dgy);
    mulmKernel<<<dimGrid, dimBlock>>>(this->d_v1, this->d_v2, this->d_v3, this->rows, this->cols);
    hipDeviceSynchronize();
}


/////////////////////////////////////////////////////////////////
// Kernel - dodawania stałej
/////////////////////////////////////////////////////////////////
template<typename T>
__global__ void addsKernel(const T* d_v1, const T* d_s, T* d_v3, size_t size){
    int index = blockIdx.x * 1024 + threadIdx.y * 32 + threadIdx.x;
    if(index < size){
        d_v3[index] = d_v1[index] + *d_s;
    }
}


template<typename T>
__host__ void MatrixKernel<T>::addsMatrixKernel(){
    int dbx = 32;
    int dby = 32;
    int dgx = static_cast<int>(this->cols/dbx)+1;
    int dgy = static_cast<int>(this->rows/dby)+1;

    dim3 dimBlock(dbx, dby);
    dim3 dimGrid (dgx, dgy);
    addsKernel<<<dimGrid, dimBlock>>>(this->d_v1, this->d_s, this->d_v3, this->size);
}


/////////////////////////////////////////////////////////////////
// Kernel - odejmowania stalej
/////////////////////////////////////////////////////////////////
template<typename T>
__global__ void subsKernel(const T* d_v1, const T* d_s, T* d_v3, size_t size){
    int index = blockIdx.x * 1024 + threadIdx.y * 32 + threadIdx.x;
    if(index < size){
        d_v3[index] = d_v1[index] - *d_s;
    }
}


template<typename T>
__host__ void MatrixKernel<T>::subsMatrixKernel(){
    int dbx = 32;
    int dby = 32;
    int dgx = static_cast<int>(this->cols/dbx)+1;
    int dgy = static_cast<int>(this->rows/dby)+1;

    dim3 dimBlock(dbx, dby);
    dim3 dimGrid (dgx, dgy);
    subsKernel<<<dimGrid, dimBlock>>>(this->d_v1, this->d_s, this->d_v3, this->size);
}


/////////////////////////////////////////////////////////////////
// Kernel - mnozenia stalej
/////////////////////////////////////////////////////////////////
template<typename T>
__global__ void mulsKernel(const T* d_v1, const T* d_s, T* d_v3, size_t size){
    int index = blockIdx.x * 1024 + threadIdx.y * 32 + threadIdx.x;
    if(index < size){
        d_v3[index] = d_v1[index] * *d_s;
    }
}


template<typename T>
__host__ void MatrixKernel<T>::mulsMatrixKernel(){
    int dbx = 32;
    int dby = 32;
    int dgx = static_cast<int>(this->cols/dbx)+1;
    int dgy = static_cast<int>(this->rows/dby)+1;    

    dim3 dimBlock(dbx, dby);
    dim3 dimGrid (dgx, dgy);
    mulsKernel<<<dimGrid, dimBlock>>>(this->d_v1, this->d_s, this->d_v3, this->size);
    hipDeviceSynchronize();
}


/////////////////////////////////////////////////////////////////
// Kernel - transpozycji
/////////////////////////////////////////////////////////////////
template<typename T>
__global__ void transposeKernel(const T* d_v1, T* d_v3, const int r, const int c){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if(row < r && col < c){
        d_v3[col * r + row] = d_v1[row * r + col];
    }
}


template<typename T>
__host__ void MatrixKernel<T>::transposeMatrixKernel(){
    int dbx = 32;
    int dby = 32;
    int dgx = static_cast<int>(this->cols/dbx)+1;
    int dgy = static_cast<int>(this->rows/dby)+1;    

    dim3 dimBlock(dbx, dby);
    dim3 dimGrid (dgx, dgy);
    transposeKernel<<<dimGrid, dimBlock>>>(this->d_v1, this->d_v3, this->rows, this->cols);
    hipDeviceSynchronize();
}


/////////////////////////////////////////////////////////////////
// przypisanie wyniku do hosta
/////////////////////////////////////////////////////////////////
template<typename T>
void MatrixKernel<T>::get_d_v3(T * h_v3){
    hipMemcpy(h_v3, d_v3, this->size * sizeof(T), hipMemcpyDeviceToHost);
}


